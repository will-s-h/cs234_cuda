#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#define TILE_SIZE 16

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_FLOAT(x) TORCH_CHECK(x.scalar_type() == torch::kFloat32, #x " must be a float32 tensor")

__global__ void matmul_tiled_kernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float C_value = 0.0f;

    for (int m = 0; m < (N + TILE_SIZE - 1) / TILE_SIZE; ++m) {
        // Load tiles into shared memory
        if (row < N && m * TILE_SIZE + tx < N)
            As[ty][tx] = A[row * N + m * TILE_SIZE + tx];
        else
            As[ty][tx] = 0.0f;

        if (col < N && m * TILE_SIZE + ty < N)
            Bs[ty][tx] = B[(m * TILE_SIZE + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < TILE_SIZE; ++k) {
            C_value += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write the result
    if (row < N && col < N)
        C[row * N + col] = C_value;
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_FLOAT(A);
    CHECK_FLOAT(B);

    TORCH_CHECK(A.dim() == 2 && A.size(0) == A.size(1), "A must be a square matrix");
    TORCH_CHECK(B.dim() == 2 && B.size(0) == B.size(1), "B must be a square matrix");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of the same size");

    int64_t N = A.size(0);

    auto C = torch::zeros({N, N}, A.options());

    const float* A_data = A.data_ptr<float>();
    const float* B_data = B.data_ptr<float>();
    float* C_data = C.data_ptr<float>();

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matmul_tiled_kernel<<<blocksPerGrid, threadsPerBlock>>>(A_data, B_data, C_data, N);

    // Check for kernel launch errors
    C10_CUDA_CHECK(hipGetLastError());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication kernel (CUDA)");
}